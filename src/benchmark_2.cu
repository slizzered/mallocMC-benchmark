#include "hip/hip_runtime.h"
/*
  mallocMC: Memory Allocator for Many Core Architectures.
  https://www.hzdr.de/crp

  Copyright 2014 Institute of Radiation Physics,
                 Helmholtz-Zentrum Dresden - Rossendorf

  Author(s):  Carlchristian Eckert - c.eckert ( at ) hzdr.de

  Permission is hereby granted, free of charge, to any person obtaining a copy
  of this software and associated documentation files (the "Software"), to deal in the Software without restriction, including without limitation the rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included in
  all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
  THE SOFTWARE.
*/

#define MALLOCMC 5
#define CUDAMALLOC 7
#define SCATTERALLOC 13
#define ALLOC_LOG 0
#define ALLOC_LIN 1

#define BENCHMARK_VERIFY 0

#include "print_machine_readable.hpp"
#include "dout.hpp"
#include "cmd_line.hpp"
#include "macros.hpp"

#include <boost/mpl/int.hpp>
#include <boost/mpl/bool.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <stdio.h>
#include <typeinfo>
#include <vector>
#include <string>
#include <utility>
#include <hiprand/hiprand_kernel.h>
#include <map>
#include <algorithm>

#if BENCHMARK_ALLOCATOR == MALLOCMC
// basic files for mallocMC
#include <mallocMC/mallocMC_overwrites.hpp>
#include <mallocMC/mallocMC_utils.hpp>
#include "benchmark_1.config.hpp"
MALLOCMC_SET_ALLOCATOR_TYPE(ScatterAllocator)
#endif


#if BENCHMARK_ALLOCATOR == SCATTERALLOC
typedef unsigned uint32;
//set the template arguments using HEAPARGS
// pagesize ... byter per page
// accessblocks ... number of superblocks
// regionsize ... number of regions for meta data structur
// wastefactor ... how much memory can be wasted per alloc (multiplicative factor)
// use_coalescing ... combine memory requests of within each warp
// resetfreedpages ... allow pages to be reused with a different size
#define HEAPARGS 4096, 8, 16, 2, true, false
#include "tools/heap_impl.cuh"
#include "tools/utils.h"
#endif






typedef std::map<int,std::map<int,std::vector<unsigned long long> > > benchmarkMap;

bool run_benchmark_2(const size_t, const unsigned, const bool, const unsigned);
std::string writeBenchmarkData();
std::string writeAveragedValues(benchmarkMap &);

std::vector<std::pair<std::string,std::string> > machine_output;


int main(int argc, char** argv){
  bool correct          = false;
  bool machine_readable = false;
  size_t heapInMB       = heapInMB_default;
  unsigned threads      = threads_default;
  unsigned blocks       = blocks_default;
  unsigned device       = 0;


  parse_cmdline(argc, argv, &heapInMB, &threads, &blocks, &machine_readable, &device);

  hipSetDevice(device);
  hipDeviceReset();
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  if( deviceProp.major < 2 ) {
    std::cerr << "Error: Compute Capability >= 2.0 required. (is ";
    std::cerr << deviceProp.major << "."<< deviceProp.minor << ")" << std::endl;
    return 1;
  }

  correct = run_benchmark_2(heapInMB, threads, machine_readable, device);
  std::cerr << threads << "    " << writeBenchmarkData() << std::endl;

  hipDeviceReset();

  if(!machine_readable || verbose){
    if(correct){
      std::cout << "\033[0;32mverification successful ✔\033[0m" << std::endl;
      return 0;
    }else{
      std::cerr << "\033[0;31mverification failed\033[0m" << std::endl;
      return 1;
    }
  }
}

__device__ int globalSuccess = 1;
__device__ int globalAllocationsInit = 0;
__device__ long long unsigned globalAllocationsContinued = 0;
__device__ long long unsigned globalFreeContinued = 0;
__device__ int globalFreeTeardown = 0;
__device__ int globalFailsInit = 0;
__device__ int globalFailsContinued = 0;
__device__ long long unsigned globalAllocClocks = 0;
__device__ long long unsigned globalFreeClocks = 0;


__global__ void cleanup_kernel(){
  printf("cleanup kernel ran successfully\n");
  globalSuccess = 1;
  globalAllocationsInit = 0;
  globalAllocationsContinued = 0;
  globalFreeContinued = 0;
  globalFreeTeardown = 0;
  globalFailsInit = 0;
  globalFailsContinued = 0;
  globalAllocClocks = 0;
  globalFreeClocks = 0;
}


void init_kernel(){
  CUDA_CHECK_KERNEL_SYNC(cleanup_kernel<<<1,1>>>());
  hipDeviceSynchronize();
}

__global__ void getBenchmarkData(
        //int *devAllocationsInit,
        long long unsigned *devAllocationsContinued,
        long long unsigned *devFreeContinued,
        //int *devFreeTeardown,
        //int *devFailsInit,
        //int *devFailsContinued,
        long long unsigned *devAllocClocks,
        long long unsigned *devFreeClocks
        ){

        //*devAllocationsInit = globalAllocationsInit;
        *devAllocationsContinued = globalAllocationsContinued;
        *devFreeContinued = globalFreeContinued;
        //*devFreeTeardown = globalFreeTeardown;
        //*devFailsInit = globalFailsInit;
        //*devFailsContinued = globalFailsContinued;
        *devAllocClocks = globalAllocClocks;
        *devFreeClocks = globalFreeClocks;
}

__global__ void getTeardown(){
  printf("Free-operations during Teardown: %d\n",globalFreeTeardown);
  printf("Total allocations: %d\n",globalAllocationsInit+globalAllocationsContinued);
  printf("Total free:        %d\n",globalFreeContinued+globalFreeTeardown);
  printf("Average clocks per alloc: %llu\n",(long long unsigned)(globalAllocClocks/globalAllocationsContinued));
  printf("Average clocks per free : %llu\n",(long long unsigned)(globalFreeClocks/globalFreeContinued));
}
__global__ void getWarmupStats(){
  printf("Alloc-operations during Warmup: %d (%d fails)\n",globalAllocationsContinued,globalFailsContinued);
  printf("Free-operations during Warmup: %d\n",globalFreeContinued);
  printf("Average clocks per alloc Warmup: %llu\n",(long long unsigned)(globalAllocClocks/globalAllocationsContinued));
  printf("Average clocks per free Warmup: %llu\n",(long long unsigned)(globalFreeClocks/globalFreeContinued));
}
__global__ void getContinuedStats(){
  printf("Alloc-operations during run: %d (%d fails)\n",globalAllocationsContinued,globalFailsContinued);
  printf("Free-operations during run: %d\n",globalFreeContinued);
  printf("Average clocks per alloc run: %llu\n",(long long unsigned)(globalAllocClocks/globalAllocationsContinued));
  printf("Average clocks per free run: %llu\n",(long long unsigned)(globalFreeClocks/globalFreeContinued));
}

/**
 * produces a uniform distribution of values from {16,32,64,128}
 */
__device__ int getAllocSizeLinScale(const int id, hiprandState_t* randomState){
  //pick a number from {0,1,2,3} (uniform distribution)
  int multiplier = ceil(hiprand_uniform(&randomState[id])*3)-1;
  return 16 << multiplier;
}

/**
 * produces a uniform distribution of values from {16,32,64,128}
 */
//__host__ int getAllocSizeLinScale(std::default_random_engine generator, std::uniform_real_distribution<float> distribution){
__host__ int getAllocSizeLinScale(){
  //pick a number from {0,1,2,3} (uniform distribution)
  //int multiplier = distribution(generator); 
  float probability = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
  int multiplier = probability*32;
  return 16 << multiplier%4;
}

/**
 * produces a logarithmic distribution of values from {16,32,64,128}
 * 64 is twice as likely as 128
 * 32 is twice as likely as 64
 * 16 is twice as likely as 32
 */
__device__ int getAllocSizeLogScale(const int id, hiprandState_t* randomState){
  //pick a number from (1,16] (uniformly distributed)
  float x = hiprand_uniform(&randomState[id])*15 + 1; 
  //get a number from {1,2,3,4}
  //picking 2 is 2 times more probable than picking 1
  //picking 3 is 4 times more probable than picking 1
  //picking 4 is 8 times more probable than picking 1
  int shift = ceil(log2(x));
#if BENCHMARK_VERIFY == 1
  assert(shift > 0);
  assert(shift <= 4);
#endif
  return 256 >> shift;
}

/**
 * produces a logarithmic distribution of values from {16,32,64,128}
 * 64 is twice as likely as 128
 * 32 is twice as likely as 64
 * 16 is twice as likely as 32
 */
//__host__ int getAllocSizeLogScale(std::default_random_engine generator, std::uniform_real_distribution<float> distribution){
__host__ int getAllocSizeLogScale(){
  //pick a number from [2,32) (uniformly distributed)
  //float x = distribution(generator); 
  float probability = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
  float x = probability * 30 + 2;
  //get a number from {1,2,3,4}
  //picking 2 is 2 times more probable than picking 1
  //picking 3 is 4 times more probable than picking 1
  //picking 4 is 8 times more probable than picking 1
  int shift = floor(log2(x));
#if BENCHMARK_VERIFY == 1
  assert(shift > 0);
  assert(shift <= 4);
#endif
  return 256 >> shift;
}

//__host__ int getAllocSize(std::default_random_engine generator, std::uniform_real_distribution<float> distribution){
__host__ int getAllocSize(){
#if BENCHMARK_ALLOCATION_SIZE == ALLOC_LOG
      //return getAllocSizeLogScale(generator, distribution);
      return getAllocSizeLogScale();
#endif
#if BENCHMARK_ALLOCATION_SIZE == ALLOC_LIN
      //return getAllocSizeLinScale(generator, distribution);
      return getAllocSizeLinScale();
#endif
#if BENCHMARK_ALLOCATION_SIZE > 7
      return BENCHMARK_ALLOCATION_SIZE;
#endif
}

__device__ int getAllocSize(const int id, hiprandState_t* randomState){
#if BENCHMARK_ALLOCATION_SIZE == ALLOC_LOG
    return getAllocSizeLogScale(id, randomState);
#endif
#if BENCHMARK_ALLOCATION_SIZE == ALLOC_LIN
    return getAllocSizeLinScale(id, randomState);
#endif
#if BENCHMARK_ALLOCATION_SIZE > 7
  return BENCHMARK_ALLOCATION_SIZE;
#endif
}

__device__ void* allocUnderTest(size_t size,long long unsigned* duration){
  long long unsigned start_time = clock64();
#if BENCHMARK_ALLOCATOR == MALLOCMC
    void* p = mallocMC::malloc(size);
#endif
#if BENCHMARK_ALLOCATOR == CUDAMALLOC
    void* p = malloc(size);
#endif
#if BENCHMARK_ALLOCATOR == SCATTERALLOC
    void* p = theHeap.alloc(size);
#endif
  long long unsigned stop_time = clock64();
  *duration = stop_time-start_time;
  return p;
}

__device__ void freeUnderTest(void* p,long long unsigned* duration){
  long long unsigned start_time = clock64();
#if BENCHMARK_ALLOCATOR == MALLOCMC
    mallocMC::free(p);
#endif
#if BENCHMARK_ALLOCATOR == CUDAMALLOC
    free(p);
#endif
#if BENCHMARK_ALLOCATOR == SCATTERALLOC
    theHeap.dealloc(p);
#endif
  long long unsigned stop_time = clock64();
  *duration = stop_time-start_time;
}



__device__ int* testRequest(int id, int alloc_size, int* p){
#if BENCHMARK_ALLOCATOR == MALLOCMC
#if BENCHMARK_VERIFY == 1
  if(p==NULL){
    int slotsRemaining = mallocMC::getAvailableSlots(alloc_size);
    if(slotsRemaining>10){
      printf("thread %d wants to allocate %d bytes (%u slots remaining), but did NOT get anything!\n",
          id, alloc_size, slotsRemaining);
      atomicAnd(&globalSuccess,0);
    }
  }
#endif
#endif
  return p;
}

__global__ void createPointerStorageInThreads(
    int*** pointerStore,
    size_t maxPointersPerThread,
    int desiredThreads,
    int* fillLevelsPerThread,
    int* pointersPerThread
    ){
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id >= desiredThreads) return;
  int** p = (int**) malloc(sizeof(int*) * maxPointersPerThread);
  if(p == NULL) atomicAnd(&globalSuccess, 0);
  pointerStore[id] = p;
  fillLevelsPerThread[id] = 0;
  pointersPerThread[id] = 0;
  //hiprand_init(seed, id, 0, &randomState[id]);
}

__global__ void allocKernel(
    int*** pointerStore,
    int maxBytesPerThread,
    int desiredThreads,
    int* fillLevelsPerThread,
    int* pointersPerThread,
    const int alloc_size
    ){
    
  const int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id >= desiredThreads) return;
  if(fillLevelsPerThread[id]+128 >= maxBytesPerThread) return;

  long long unsigned duration=0llu;
  int * p = (int*) allocUnderTest(alloc_size, &duration);

  if(p == NULL){
    atomicAdd(&globalFailsInit,1);
  }
  else{
    p[0] = alloc_size;
    fillLevelsPerThread[id] += alloc_size;
    pointerStore[id][pointersPerThread[id]++] = p;
    atomicAdd(&globalAllocationsContinued,1);
    atomicAdd(&globalAllocClocks, duration);
  }
}


__global__ void freeKernel(
    int*** pointerStore,
    int desiredThreads,
    int* fillLevelsPerThread,
    int* pointersPerThread,
    const int alloc_size
    ){
    
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id >= desiredThreads) return;
  if(pointersPerThread[id] == 0) return;

  int free_size = pointerStore[id][--pointersPerThread[id]][0];
  //int free_size = alloc_size;
  long long unsigned duration=0llu;

  freeUnderTest(pointerStore[id][pointersPerThread[id]],&duration);

  fillLevelsPerThread[id] -= free_size;
  atomicAdd(&globalFreeContinued,1llu);
  atomicAdd(&globalFreeClocks, duration);
}



__global__ void deallocAll(
    int*** pointerStore,
    int desiredThreads,
    int* fillLevelsPerThread,
    int* pointersPerThread
    ){
    
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id >= desiredThreads) return;

  int fillLevel = fillLevelsPerThread[id];
  int** pointerStoreReg = pointerStore[id];
  int pointersPerThreadReg = pointersPerThread[id];


  while(pointersPerThreadReg > 0) { 
    int free_size = pointerStoreReg[--pointersPerThreadReg][0];
    long long unsigned duration=0llu;
    freeUnderTest(pointerStoreReg[pointersPerThreadReg],&duration);
    fillLevel -= free_size;
    atomicAdd(&globalFreeTeardown,1);
  }  

  free(pointerStore[id]);
}

__global__ void getSuccessState(int* success){
  printf("Allocations done during initialization: %d (%d times, no memory was available)\n",
      globalAllocationsInit,globalFailsInit);
  printf("Allocations done while running: %llu (%d times, no memory was available)\n",
      globalAllocationsContinued,globalFailsContinued);
  printf("Free-operations done while running: %llu\n",
      globalFreeContinued);
  success[0] = globalSuccess;
}

std::string writeBenchmarkData(){
  //int hostAllocationsInit = 0;
  long long unsigned hostAllocationsContinued = 0;
  long long unsigned hostFreeContinued = 0;
  //int hostFreeTeardown = 0;
  //int hostFailsInit = 0;
  //int hostFailsContinued = 0;
  long long unsigned hostAllocClocks = 0;
  long long unsigned hostFreeClocks = 0;
  //int *devAllocationsInit;
  long long unsigned *devAllocationsContinued;
  long long unsigned *devFreeContinued;
  //int *devFreeTeardown;
  //int *devFailsInit;
  //int *devFailsContinued;
  long long unsigned *devAllocClocks;
  long long unsigned *devFreeClocks;

  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &devAllocationsContinued,sizeof(long long unsigned)));
  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &devFreeContinued,sizeof(long long unsigned)));
  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &devAllocClocks,sizeof(long long unsigned)));
  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &devFreeClocks,sizeof(long long unsigned)));
  CUDA_CHECK_KERNEL_SYNC(getBenchmarkData<<<1,1>>>(
        //devAllocationsInit,
        devAllocationsContinued,
        devFreeContinued,
        //devFreeTeardown,
        //devFailsInit,
        //devFailsContinued,
        devAllocClocks,
        devFreeClocks
        ));
  BENCHMARK_CHECKED_CALL(hipMemcpy(&hostAllocationsContinued,devAllocationsContinued,sizeof(long long unsigned),hipMemcpyDeviceToHost));
  BENCHMARK_CHECKED_CALL(hipMemcpy(&hostFreeContinued,devFreeContinued,sizeof(long long unsigned),hipMemcpyDeviceToHost));
  BENCHMARK_CHECKED_CALL(hipMemcpy(&hostAllocClocks,devAllocClocks,sizeof(long long unsigned),hipMemcpyDeviceToHost));
  BENCHMARK_CHECKED_CALL(hipMemcpy(&hostFreeClocks,devFreeClocks,sizeof(long long unsigned),hipMemcpyDeviceToHost));

  std::stringstream ss;
  ss << hostAllocClocks/hostAllocationsContinued << "    ";
  ss << hostFreeClocks/hostFreeContinued;

  return ss.str();
}


/**
 * Verify the heap allocation of mallocMC
 *
 * Allocates as much memory as the heap allows. Make sure that allocated
 * memory actually holds the correct values without corrupting them. Will
 * fill the datastructure with values that are relative to the index and
 * later evalute, if the values inside stayed the same after allocating all
 * memory.
 * Datastructure: Array that holds up to nPointers pointers to arrays of size
 * ELEMS_PER_SLOT, each being of type allocElem_t.
 *
 * @return true if the verification was successful,
 *         false otherwise
 */
bool run_benchmark_2(
    const size_t heapMB,
    const unsigned desiredThreads,
    const bool machine_readable,
    const unsigned device
    ){

  int h_globalSuccess=0;
  //std::default_random_engine generator;
  //std::uniform_real_distribution<float> distribution(2,32);

  init_kernel();
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

  unsigned maxBlocksPerSM = 8;
  if(deviceProp.major > 2) maxBlocksPerSM *= 2; //set to 16 for 3.0 and higher
  if(deviceProp.major >= 5) maxBlocksPerSM *= 2; //increase again to 32 for 5.0 and higher

  //use the smallest possible blocksize that is still able to fill the multiprocessor
  const size_t threadsUsedInBlock = deviceProp.maxThreadsPerMultiProcessor / maxBlocksPerSM;
  const size_t maxUsefulBlocks = maxBlocksPerSM * deviceProp.multiProcessorCount;
  dout() << "threadsUsedInBlock: " << threadsUsedInBlock << std::endl;
  dout() << "maxUsefulBlocks:    " << maxUsefulBlocks << std::endl;
  dout() << "Clock Frequency:    " << deviceProp.clockRate/1000.0 << "MHz" << std::endl;
  
  const unsigned threads = threadsUsedInBlock;
  const unsigned blocks  = maxUsefulBlocks;

  const size_t usableMemoryMB   = deviceProp.totalGlobalMem / size_t(1024U * 1024U);
  if(heapMB > usableMemoryMB/2) dout() << "Warning: heapSize fills more than 50% of global Memory" << std::endl;

  const size_t heapSize         = size_t(1024U*1024U) * heapMB;
  machine_output.push_back(MK_STRINGPAIR(heapSize));

  //if a single thread allocates only the minimal chunksize, it can not exceed this number
  size_t maxStoredChunks = heapMB * size_t(1024U * 1024U) / size_t(16U);
  size_t maxMemPerThread = heapMB * size_t(1024U * 1024U) / desiredThreads;
  int maxChunksPerThread = maxMemPerThread / 16;
  int maxChunksTotal = maxChunksPerThread * desiredThreads;

  int*** pointerStoreForThreads;
  int* fillLevelsPerThread;
  int* pointersPerThread;
  hiprandState_t* randomState;
  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &pointerStoreForThreads, desiredThreads*sizeof(int**)));
  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &fillLevelsPerThread, desiredThreads*sizeof(int)));
  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &pointersPerThread, desiredThreads*sizeof(int)));
  BENCHMARK_CHECKED_CALL(hipMalloc((void**) &randomState, desiredThreads * sizeof(hiprandState_t)));



  //dout() << "maxStoredChunks: " << maxStoredChunks << std::endl;
  size_t pointerSize = maxChunksTotal*sizeof(int**)*4;
  dout() << "necessary memory for pointers: " << pointerSize << std::endl;
  dout() << "reserved Heapsize:             " << heapSize << std::endl;

#if BENCHMARK_ALLOCATOR == MALLOCMC
    hipDeviceSetLimit(hipLimitMallocHeapSize, pointerSize);
    mallocMC::initHeap(heapSize);
#endif
#if BENCHMARK_ALLOCATOR == CUDAMALLOC
    hipDeviceSetLimit(hipLimitMallocHeapSize, pointerSize + heapSize);
#endif
#if BENCHMARK_ALLOCATOR == SCATTERALLOC
    hipDeviceSetLimit(hipLimitMallocHeapSize, pointerSize);
    initHeap(heapSize);
#endif

  size_t maxPointersPerThread = ceil(float(maxStoredChunks)/desiredThreads);
  CUDA_CHECK_KERNEL_SYNC(createPointerStorageInThreads<<<blocks,threads>>>(
      pointerStoreForThreads,
      maxChunksPerThread,
      desiredThreads,
      fillLevelsPerThread,
      pointersPerThread
      ));

  
#if BENCHMARK_ALLOCATOR != CUDAMALLOC
    for(int i=16;i<256;i = i << 1){
      if(BENCHMARK_ALLOCATION_SIZE > 1) i = BENCHMARK_ALLOCATION_SIZE;
#if BENCHMARK_ALLOCATOR == MALLOCMC
      dout() << "before warmup: free slots of size " << i << ": " << mallocMC::getAvailableSlots(i) << std::endl;
#else 
      dout() << "before warmup: free slots of size " << i << ": " << getAvailableSlotsHost(i) << std::endl;
#endif
      if(BENCHMARK_ALLOCATION_SIZE > 1) break;
    }
#endif

    srand(31415);
    for(int warm_i=0; warm_i < 50000; ++warm_i){
      float probability = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
      //int alloc_size = getAllocSize(generator,distribution);
      int alloc_size = getAllocSize();
      if(probability <= 0.75){
        CUDA_CHECK_KERNEL_SYNC(allocKernel<<<blocks,threads>>>(
              pointerStoreForThreads,
              maxMemPerThread,
              desiredThreads,
              fillLevelsPerThread,
              pointersPerThread,
              alloc_size
              ));
      }

      probability = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
      if(probability <= 0.75){
        CUDA_CHECK_KERNEL_SYNC(freeKernel<<<blocks,threads>>>(
              pointerStoreForThreads,
              desiredThreads,
              fillLevelsPerThread,
              pointersPerThread,
              alloc_size
              ));

      }
    }
    hipDeviceSynchronize();
    CUDA_CHECK_KERNEL_SYNC(getWarmupStats<<<1,1>>>());
    CUDA_CHECK_KERNEL_SYNC(cleanup_kernel<<<1,1>>>());
    dout() << "WARMUP COMPLETE" << std::endl;

    for(int cont_i=0; cont_i < 50000; ++cont_i){
      float probability = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
      //int alloc_size = getAllocSize(generator,distribution);
      int alloc_size = getAllocSize();
      if(probability <= 0.75){
        CUDA_CHECK_KERNEL_SYNC(allocKernel<<<blocks,threads>>>(
              pointerStoreForThreads,
              maxMemPerThread,
              desiredThreads,
              fillLevelsPerThread,
              pointersPerThread,
              alloc_size
              ));
      }

      probability = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
      if(probability <= 0.75){
        CUDA_CHECK_KERNEL_SYNC(freeKernel<<<blocks,threads>>>(
              pointerStoreForThreads,
              desiredThreads,
              fillLevelsPerThread,
              pointersPerThread,
              alloc_size
              ));
      }
    }


  int* d_success;
  hipMalloc((void**) &d_success,sizeof(int));
  getSuccessState<<<1,1>>>(d_success);
  BENCHMARK_CHECKED_CALL(hipMemcpy((void*) &h_globalSuccess,d_success, sizeof(int), hipMemcpyDeviceToHost));
  machine_output.push_back(MK_STRINGPAIR(h_globalSuccess));
//  print_machine_readable(machine_output);

  // release all memory
  CUDA_CHECK_KERNEL_SYNC(deallocAll<<<blocks,threads>>>(
      pointerStoreForThreads,
      desiredThreads,
      fillLevelsPerThread,
      pointersPerThread
      ));
  CUDA_CHECK_KERNEL_SYNC(getTeardown<<<1,1>>>());
  hipDeviceSynchronize();

#if BENCHMARK_ALLOCATOR != CUDAMALLOC
    for(int i=16;i<256;i = i << 1){
      if(BENCHMARK_ALLOCATION_SIZE > 1) i = BENCHMARK_ALLOCATION_SIZE;
#if BENCHMARK_ALLOCATOR == MALLOCMC
      dout() << "after filling: free slots of size " << i << ": " << mallocMC::getAvailableSlots(i) << std::endl;
#else 
      dout() << "after filling: free slots of size " << i << ": " << getAvailableSlotsHost(i) << std::endl;
#endif
      if(BENCHMARK_ALLOCATION_SIZE > 1) break;
    }
#endif



#if BENCHMARK_ALLOCATOR == MALLOCMC
    h_globalSuccess = h_globalSuccess && (mallocMC::getAvailableSlots(16)==1036320);
    mallocMC::finalizeHeap();
#endif
#if BENCHMARK_ALLOCATOR == SCATTERALLOC
    h_globalSuccess = h_globalSuccess && (getAvailableSlotsHost(16)==1036320);
#endif
  hipFree(d_success);
  hipFree(pointerStoreForThreads);
  hipFree(fillLevelsPerThread);
  hipFree(pointersPerThread);
  hipFree(randomState);
  dout() << "done "<< std::endl;

  return h_globalSuccess;
}
